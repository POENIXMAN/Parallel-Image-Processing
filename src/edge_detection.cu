#include <stdio.h>
#include <stdlib.h>
#include <png.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>

typedef struct
{
    int height;
    int width;
    int pixel_size;
    png_infop info_ptr;
    png_byte *buf;
} PNG_RAW;

long long timeInMilliseconds(void)
{
    struct timeval tv;

    gettimeofday(&tv, NULL);
    return (((long long)tv.tv_sec) * 1000) + (tv.tv_usec / 1000);
}

PNG_RAW *read_png(char *file_name)
{
    PNG_RAW *png_raw = (PNG_RAW *)malloc(sizeof(PNG_RAW));

    FILE *fp = fopen(file_name, "rb");
    png_structp png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_read_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_bytepp row_pointers = png_get_rows(png_ptr, info_ptr);
    int width = png_get_image_width(png_ptr, info_ptr);
    int height = png_get_image_height(png_ptr, info_ptr);
    int pixel_size = png_get_rowbytes(png_ptr, info_ptr) / width;
    png_raw->width = width;
    png_raw->height = height;
    png_raw->pixel_size = pixel_size;
    png_raw->buf = (png_byte *)malloc(width * height * pixel_size * sizeof(png_byte));
    png_raw->info_ptr = info_ptr;
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            png_raw->buf[k++] = row_pointers[i][j];
        }
    png_destroy_read_struct(&png_ptr, NULL, NULL);
    fclose(fp);
    return png_raw;
}

void write_png(char *file_name, PNG_RAW *png_raw)
{
    FILE *fp = fopen(file_name, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_init_io(png_ptr, fp);
    png_infop info_ptr = png_raw->info_ptr;
    int width = png_raw->width;
    int height = png_raw->height;
    int pixel_size = png_raw->pixel_size;
    png_bytepp row_pointers;
    row_pointers = (png_bytepp)malloc(height * sizeof(png_bytep));
    for (int i = 0; i < height; i++)
        row_pointers[i] = (png_bytep)malloc(width * pixel_size);
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            row_pointers[i][j] = png_raw->buf[k++];
        }

    png_set_rows(png_ptr, info_ptr, row_pointers);
    png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    for (int i = 0; i < height; i++)
        free(row_pointers[i]);
    free(row_pointers);
    fclose(fp);
}


/**
 * CUDA kernel for Sobel edge detection operation.
 *
 * This kernel computes the gradient magnitude of each pixel in the input image using
 * the Sobel filter kernels. It updates the pixel values in-place with the grayscale
 * representation of the gradient magnitude.
 *
 * @param d_P         Pointer to the input image data in device memory.
 * @param height      Height of the input image.
 * @param width       Width of the input image.
 * @param pixel_size  Number of bytes per pixel.
 */
__global__ void SobelKernel(png_byte *d_P, int height, int width, int pixel_size)
{
    // Define the Sobel filter kernels
    int Gx[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int Gy[3][3] = {{-1, -2, -1}, {0, 0, 0}, {1, 2, 1}};

    // Calculate the global index of the current thread
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = tid_y * width + tid_x;

    // Check if the thread is within the image boundaries
    if (tid_x >= width || tid_y >= height)
    {
        return;
    }

   // Initialize variables to accumulate gradients in the x and y directions
int sumx = 0, sumy = 0;

// Iterate over a 3x3 neighborhood of pixels centered around the current thread's position
for (int i = -1; i <= 1; i++)
{
    for (int j = -1; j <= 1; j++)
    {
        // Calculate the coordinates of the pixel in the neighborhood
        int y = tid_y + i;
        int x = tid_x + j;

        // Check if the pixel coordinates are within the image boundaries
        if (x >= 0 && x < width && y >= 0 && y < height)
        {
            // Compute the index of the pixel in the input image buffer
            int index = (y * width + x) * pixel_size;

            // Retrieve the red, green, and blue color values of the pixel
            int r = d_P[index];
            int g = d_P[index + 1];
            int b = d_P[index + 2];

            // Calculate the luminance value of the pixel using the RGB values
            float luminance_value = 0.2126f * r + 0.7152f * g + 0.0722f * b;

            // Accumulate the gradients in the x and y directions
            sumx += Gx[i + 1][j + 1] * luminance_value;
            sumy += Gy[i + 1][j + 1] * luminance_value;
        }
    }
}


    // Calculate the gradient magnitude of the current pixel
    float gradient_magnitude = sqrtf((float)(sumx * sumx + sumy * sumy));

    // Set the color of the pixel based on the gradient magnitude
    png_byte gray = (png_byte)(gradient_magnitude * 255.0f / sqrtf(2.0f) / 255.0f);
    d_P[tid * 3] = gray;
    d_P[tid * 3 + 1] = gray;
    d_P[tid * 3 + 2] = gray;
}

 

void process_on_device(PNG_RAW *png_raw)
{

    // assume that the picture is m × n,
    // m pixels in y dimension and n pixels in x dimension
    // input d_Pin has been allocated on and copied to device
    // output d_Pout has been allocated on device
    int m = png_raw->height;
    int n = png_raw->width;
    int pixel_size = png_raw->pixel_size;

    dim3 DimGrid((n - 1) / 16 + 1, (m - 1) / 16 + 1, 1);
    dim3 DimBlock(16, 16, 1);

    png_byte *d_P;
    hipError_t err;

    long long start = timeInMilliseconds();

    err = hipMalloc((void **)&d_P, m * n * pixel_size * sizeof(png_byte));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_P, png_raw->buf, m * n * pixel_size, hipMemcpyHostToDevice);

    SobelKernel<<<DimGrid, DimBlock>>>(d_P, m, n, pixel_size);

    hipMemcpy(png_raw->buf, d_P, m * n * pixel_size, hipMemcpyDeviceToHost);

    long long end = timeInMilliseconds();

    printf("timing on Device is %lld millis\n", end - start);
}



/**
 * Perform image processing operations on the host (CPU).
 *
 * This function computes the luminance and edge detection values for each pixel
 * in the given PNG image data structure. It modifies the image data in-place by
 * updating the RGB values with the computed edge values.
 *
 * @param png_raw A pointer to the PNG_RAW structure representing the input image.
 */
void process_on_host(PNG_RAW *png_raw)
{
    // Start timing
    long long start = timeInMilliseconds();

    // Get the width and height of the image
    int width = png_raw->width;
    int height = png_raw->height;

    // Create a temporary buffer to store the luminance values
    int *luminance = (int *)malloc(sizeof(int) * width * height);

    // Compute the luminance values for each pixel
    for (int i = 0; i < width * height; i++)
    {
        int r = png_raw->buf[i * 3];
        int g = png_raw->buf[i * 3 + 1];
        int b = png_raw->buf[i * 3 + 2];

        // Compute the luminance value using the ITU-R BT.709 coefficients
        int luminance_value = (2126 * r + 7152 * g + 722 * b) / 10000;
        luminance[i] = luminance_value;
    }

    // Compute the edge detection values for each pixel
    for (int y = 1; y < height - 1; y++)
    {
        for (int x = 1; x < width - 1; x++)
        {
            int i = y * width + x;

            // Compute the gradients in the x and y directions using Sobel operator
            int gx = -luminance[i - width - 1] - 2 * luminance[i - 1] - luminance[i + width - 1] +
                     luminance[i - width + 1] + 2 * luminance[i + 1] + luminance[i + width + 1];
            int gy = -luminance[i - width - 1] - 2 * luminance[i - width] - luminance[i - width + 1] +
                     luminance[i + width - 1] + 2 * luminance[i + width] + luminance[i + width + 1];

            // Compute the edge value using the Euclidean norm of the gradients
            int edge_value = (int)(sqrt(gx * gx + gy * gy) * 255.0 / (sqrt(2.0) * 255.0) + 0.5);

            // Clamp the edge value to the range [0, 255]
            edge_value = edge_value < 0 ? 0 : (edge_value > 255 ? 255 : edge_value);

            // Update the RGB values with the computed edge value
            png_raw->buf[i * 3] = (png_byte)edge_value;
            png_raw->buf[i * 3 + 1] = (png_byte)edge_value;
            png_raw->buf[i * 3 + 2] = (png_byte)edge_value;
        }
    }

    // Free the temporary luminance buffer
    free(luminance);

    // End timing and print the elapsed time
    long long end = timeInMilliseconds();
    printf("Timing on host: %lld milliseconds\n",end - start);
}



int main(int argc, char **argv)
{
    int on_host = 2;

    if (argv[3] != NULL && strcmp(argv[3], "-d") == 0)
        on_host = 0;

    PNG_RAW *png_raw = read_png(argv[1]);
    if (png_raw->pixel_size != 3)
    {
        printf("Error, png file must be on 3 Bytes per pixel\n");
        exit(0);
    }
    else
        printf("RGB Processing for Image of %d x %d pixels\n", png_raw->width, png_raw->height);

    if (on_host == 1)
        process_on_host(png_raw);
    else
        process_on_device(png_raw);

    write_png(argv[2], png_raw);

    printf("Processing finished \n");
}
