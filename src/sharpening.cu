#include <stdio.h>
#include <stdlib.h>
#include <png.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
typedef struct
{
    int height;
    int width;
    int pixel_size;
    png_infop info_ptr;
    png_byte *buf;
} PNG_RAW;

long long timeInMilliseconds(void)
{
    struct timeval tv;

    gettimeofday(&tv, NULL);
    return (((long long)tv.tv_sec) * 1000) + (tv.tv_usec / 1000);
}

PNG_RAW *read_png(char *file_name)
{
    PNG_RAW *png_raw = (PNG_RAW *)malloc(sizeof(PNG_RAW));

    FILE *fp = fopen(file_name, "rb");
    png_structp png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_read_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_bytepp row_pointers = png_get_rows(png_ptr, info_ptr);
    int width = png_get_image_width(png_ptr, info_ptr);
    int height = png_get_image_height(png_ptr, info_ptr);
    int pixel_size = png_get_rowbytes(png_ptr, info_ptr) / width;
    png_raw->width = width;
    png_raw->height = height;
    png_raw->pixel_size = pixel_size;
    png_raw->buf = (png_byte *)malloc(width * height * pixel_size * sizeof(png_byte));
    png_raw->info_ptr = info_ptr;
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            png_raw->buf[k++] = row_pointers[i][j];
        }
    png_destroy_read_struct(&png_ptr, NULL, NULL);
    fclose(fp);
    return png_raw;
}

void write_png(char *file_name, PNG_RAW *png_raw)
{
    FILE *fp = fopen(file_name, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_init_io(png_ptr, fp);
    png_infop info_ptr = png_raw->info_ptr;
    int width = png_raw->width;
    int height = png_raw->height;
    int pixel_size = png_raw->pixel_size;
    png_bytepp row_pointers;
    row_pointers = (png_bytepp)malloc(height * sizeof(png_bytep));
    for (int i = 0; i < height; i++)
        row_pointers[i] = (png_bytep)malloc(width * pixel_size);
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            row_pointers[i][j] = png_raw->buf[k++];
        }

    png_set_rows(png_ptr, info_ptr, row_pointers);
    png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    for (int i = 0; i < height; i++)
        free(row_pointers[i]);
    free(row_pointers);
    fclose(fp);
}

__global__ void PictureKernel(png_byte *P, int m, int n)
{
    int tile_x = blockDim.x;
    int tile_y = blockDim.y;

    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = bx * tile_x + tx;
    int y = by * tile_y + ty;

    if (x < n && y < m)
    {
        int i = y * n + x;

        // Calculate the average color of neighboring pixels within a 7x7 window
        int R_sharp = 0;
        int G_sharp = 0;
        int B_sharp = 0;
        int cnt = 0;

        for (int dx = -3; dx <= 3; dx++)
        {
            for (int dy = -3; dy <= 3; dy++)
            {
                int nx = x + dx;
                int ny = y + dy;

                if (nx >= 0 && nx < n && ny >= 0 && ny < m)
                {
                    int j = ny * n + nx;
                    R_sharp += P[j * 3];
                    G_sharp += P[j * 3 + 1];
                    B_sharp += P[j * 3 + 2];
                    cnt++;
                }
            }
        }

        int R_avg = R_sharp / cnt;
        int G_avg = G_sharp / cnt;
        int B_avg = B_sharp / cnt;

        // Calculate the sharp color of the pixel by subtracting the average color from the original color
        R_sharp = P[i * 3] + (P[i * 3] - R_avg) * 2;
        G_sharp = P[i * 3 + 1] + (P[i * 3 + 1] - G_avg) * 2;
        B_sharp = P[i * 3 + 2] + (P[i * 3 + 2] - B_avg) * 2;

        // Set the pixel color to the sharp color, saturating it to avoid overflow
        P[i * 3] = min(max(R_sharp, 0), 255);
        P[i * 3 + 1] = min(max(G_sharp, 0), 255);
        P[i * 3 + 2] = min(max(B_sharp, 0), 255);
    }
}

void process_on_device(PNG_RAW *png_raw)
{

    // assume that the picture is m × n,
    // m pixels in y dimension and n pixels in x dimension
    // input d_Pin has been allocated on and copied to device
    // output d_Pout has been allocated on device
    int m = png_raw->height;
    int n = png_raw->width;
    int pixel_size = png_raw->pixel_size;

    dim3 DimGrid((n - 1) / 16 + 1, (m - 1) / 16 + 1, 1);
    dim3 DimBlock(16, 16, 1);

    png_byte *d_P;
    hipError_t err;

    long long start = timeInMilliseconds();

    err = hipMalloc((void **)&d_P, m * n * pixel_size * sizeof(png_byte));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_P, png_raw->buf, m * n * pixel_size, hipMemcpyHostToDevice);

    PictureKernel<<<DimGrid, DimBlock>>>(d_P, m, n);

    hipMemcpy(png_raw->buf, d_P, m * n * pixel_size, hipMemcpyDeviceToHost);

    long long end = timeInMilliseconds();

    printf("timing on Device is %lld millis\n", end - start);
}

int main(int argc, char **argv)
{
    printf("Starting sharpening process \n");
    PNG_RAW *png_raw = read_png(argv[1]);
    if (png_raw->pixel_size != 3)
    {
        printf("Error, png file must be on 3 Bytes per pixel\n");
        exit(0);
    }
    else
        printf("RGB Processing for Image of %d x %d pixels\n", png_raw->width, png_raw->height);

    process_on_device(png_raw);

    write_png(argv[2], png_raw);

    printf("Processing finished \n ____________________________________________________________________________________________\n");
}