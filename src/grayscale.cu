
#include <stdio.h>
#include <stdlib.h>
#include <png.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
typedef struct
{
    int height;
    int width;
    int pixel_size;
    png_infop info_ptr;
    png_byte *buf;
} PNG_RAW;

long long timeInMilliseconds(void)
{
    struct timeval tv;

    gettimeofday(&tv, NULL);
    return (((long long)tv.tv_sec) * 1000) + (tv.tv_usec / 1000);
}

PNG_RAW *read_png(char *file_name)
{
    PNG_RAW *png_raw = (PNG_RAW *)malloc(sizeof(PNG_RAW));

    FILE *fp = fopen(file_name, "rb");
    png_structp png_ptr = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop info_ptr = png_create_info_struct(png_ptr);
    png_init_io(png_ptr, fp);
    png_read_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_bytepp row_pointers = png_get_rows(png_ptr, info_ptr);
    int width = png_get_image_width(png_ptr, info_ptr);
    int height = png_get_image_height(png_ptr, info_ptr);
    int pixel_size = png_get_rowbytes(png_ptr, info_ptr) / width;
    png_raw->width = width;
    png_raw->height = height;
    png_raw->pixel_size = pixel_size;
    png_raw->buf = (png_byte *)malloc(width * height * pixel_size * sizeof(png_byte));
    png_raw->info_ptr = info_ptr;
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            png_raw->buf[k++] = row_pointers[i][j];
        }
    png_destroy_read_struct(&png_ptr, NULL, NULL);
    fclose(fp);
    return png_raw;
}

void write_png(char *file_name, PNG_RAW *png_raw)
{
    FILE *fp = fopen(file_name, "wb");
    png_structp png_ptr = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_init_io(png_ptr, fp);
    png_infop info_ptr = png_raw->info_ptr;
    int width = png_raw->width;
    int height = png_raw->height;
    int pixel_size = png_raw->pixel_size;
    png_bytepp row_pointers;
    row_pointers = (png_bytepp)malloc(height * sizeof(png_bytep));
    for (int i = 0; i < height; i++)
        row_pointers[i] = (png_bytep)malloc(width * pixel_size);
    int k = 0;
    for (int i = 0; i < height; i++)
        for (int j = 0; j < width * pixel_size; j++)
        {
            row_pointers[i][j] = png_raw->buf[k++];
        }

    png_set_rows(png_ptr, info_ptr, row_pointers);
    png_write_png(png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);
    png_destroy_write_struct(&png_ptr, &info_ptr);
    for (int i = 0; i < height; i++)
        free(row_pointers[i]);
    free(row_pointers);
    fclose(fp);
}

__global__ void PictureKernel(png_byte *d_P, int height, int width)
{
    // Calculate the global index of the current thread
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = tid_y * width + tid_x; // global index

    // Check if the thread is within the image boundaries
    if (tid_x >= width || tid_y >= height)
    {
        return;
    }

    // Calculate the grayscale value of the pixel
    int index = tid * 3; // Each pixel has 3 color components: red, green, and blue
    png_byte gray = (png_byte)(0.21f * d_P[index] + 0.72f * d_P[index + 1] + 0.07f * d_P[index + 2]);

    // Set the color of the pixel to grayscale
    d_P[index] = gray;
    d_P[index + 1] = gray;
    d_P[index + 2] = gray;
}

void process_on_host(PNG_RAW *png_raw)
{
    long long start = timeInMilliseconds();
    for (int i = 0; i < png_raw->width * png_raw->height; i++)
    {
        int luminance_value = 0.2126 * png_raw->buf[i * 3] + 0.7152 * png_raw->buf[i * 3 + 1] + 0.0722 * png_raw->buf[i * 3 + 2];
        png_raw->buf[i * 3] = (png_byte)luminance_value;
        png_raw->buf[i * 3 + 1] = (png_byte)luminance_value;
        png_raw->buf[i * 3 + 2] = (png_byte)luminance_value;
    }
    long long end = timeInMilliseconds();
    printf("timing on host is %lld millis\n", end - start);
}

void process_on_device(PNG_RAW *png_raw)
{

    // assume that the picture is m × n,
    // m pixels in y dimension and n pixels in x dimension
    // input d_Pin has been allocated on and copied to device
    // output d_Pout has been allocated on device
    int m = png_raw->height;
    int n = png_raw->width;
    int pixel_size = png_raw->pixel_size;

    dim3 DimGrid((n - 1) / 16 + 1, (m - 1) / 16 + 1, 1);
    dim3 DimBlock(16, 16, 1);

    png_byte *d_P;
    hipError_t err;

    long long start = timeInMilliseconds();

    err = hipMalloc((void **)&d_P, m * n * pixel_size * sizeof(png_byte));
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }
    hipMemcpy(d_P, png_raw->buf, m * n * pixel_size, hipMemcpyHostToDevice);

    PictureKernel<<<DimGrid, DimBlock>>>(d_P, m, n);

    hipMemcpy(png_raw->buf, d_P, m * n * pixel_size, hipMemcpyDeviceToHost);

    long long end = timeInMilliseconds();

    printf("timing on Device is %lld millis\n", end - start);
}

int main(int argc, char **argv)
{
    printf("Starting grayscale process \n");
    int on_host = 2;

    if (argv[3] != NULL && strcmp(argv[3], "-d") == 0)
        on_host = 0;

    PNG_RAW *png_raw = read_png(argv[1]);
    if (png_raw->pixel_size != 3)
    {
        printf("Error, png file must be on 3 Bytes per pixel\n");
        exit(0);
    }
    else
        printf("RGB Processing for Image of %d x %d pixels\n", png_raw->width, png_raw->height);

    if (on_host == 1)
        process_on_host(png_raw);
    else
        process_on_device(png_raw);

    write_png(argv[2], png_raw);

    printf("Processing finished \n ____________________________________________________________________________________________\n");
}
